////
//// The kernel functions for an class1
////

///
/// The include files for class1
///
#include "class1.h"
#include "class1_kernel.h"

//=================================================================================================

///
/// Run main GPU function
///
template <typename T>
int Class1<T>::runGPU( const int srcStep, const int srcCols, const int srcRows ) const
{ 

    NppStatus npp_err;

    //----------------------------------------------------------------------

    ///
    /// Calculate mean values per channel
    ///

    {

        //
        // Set variables needed for NPP functions
        //
        const NppiSize srcSize = { srcCols, srcRows };
        Npp8u * pDeviceBuffer;
        int nBufferSize;

        // Calculate buffer size of scratch buffer needed to calculate sum
        npp_err = nppiMeanGetBufferHostSize_8u_C3R( srcSize, &nBufferSize );

        // Check NPP error status
        if( npp_err != NPP_SUCCESS ) {
        std::cerr << "Error occured during nppiMeanGetBufferHostSize_8u_C3R run: "
              << "( NPP Status: " << npp_err << " ):"
              << __FILE__ << " : " << __LINE__
              << std::endl;
        exit(1);
        }

        // Allocate the scratch buffer 
        hipMalloc((void **)(&pDeviceBuffer), nBufferSize);

        //
        // Run NPP's mean function
        //
        npp_err = nppiMean_8u_C3R( d_iImage, srcStep, srcSize, pDeviceBuffer, d_chMeans );

        // Check NPP error status
        if( npp_err != NPP_SUCCESS ) {
        std::cerr << "Error occured during nppiMean_8u_C3R run: "
              << "( NPP Status: " << npp_err << " ):"
              << __FILE__ << " : " << __LINE__
              << std::endl;
        exit(1);
        }

        // Free memory for scratch buffer
        hipFree(pDeviceBuffer);

    }

    //----------------------------------------------------------------------

    return 0;

}

////
//// Explicit instantiation for the template class
////
template class Class1<float>;

